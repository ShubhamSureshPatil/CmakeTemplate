#include <iostream>
#include "gpu.hpp"
#include "hip/hip_runtime_api.h"

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 0-th device
    std::cout << deviceProp.multiProcessorCount << std::endl;
}
